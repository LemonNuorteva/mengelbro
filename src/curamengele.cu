#include "hip/hip_runtime.h"
#include "cudamengele.h"

#include <unistd.h>
#include <err.h>

__device__ uint32_t calc(
    const double r0,
	const double i0,
    const FrameParams& params
)
{
	uint32_t iterator = 0;
	real r{};
	real r2{};
	real i{};
	real i2{};

	while (r2 + i2 <= 4.0 && iterator < params.maxIters)
	{
		i = 2*x*i + i0;
		r = r2 - i2 + r0;
		r2 = r*r;
		i2 = i*i;
		iterator++;
	}

	return iterator;
}

__global__ void mandel_kernel(uint32_t *counts, double xmin, double ymin,
            double step, int max_iter, int dim, uint32_t *colors) {
    int pix_per_thread = dim * dim / (gridDim.x * blockDim.x);
    int tId = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = pix_per_thread * tId;
    for (int i = offset; i < offset + pix_per_thread; i++){
        int x = i % dim;
        int y = i / dim;
        double cr = xmin + x * step;
        double ci = ymin + y * step;
        counts[y * dim + x]  = colors[mandel_double(cr, ci, max_iter)];
    }
    if (gridDim.x * blockDim.x * pix_per_thread < dim * dim
            && tId < (dim * dim) - (blockDim.x * gridDim.x)){
        int i = blockDim.x * gridDim.x * pix_per_thread + tId;
        int x = i % dim;
        int y = i / dim;
        double cr = xmin + x * step;
        double ci = ymin + y * step;
        counts[y * dim + x]  = colors[mandel_double(cr, ci, max_iter)];
    }

}

const Frame& CuMengele::calcFrame(
	const FrameParams& fraPar,
	Frame& frame
)
{
	hipError_t err = hipSuccess;

	static uint32_t *colors;
	uint32_t *dev_colors;

	const size_t color_size = (params.maxIters) * sizeof(uint32_t);
	colors = (uint32_t *) malloc(color_size);
	hipMalloc((void**)&dev_colors, color_size);

	hipMemcpy(dev_colors, colors, color_size, hipMemcpyHostToDevice);
	free(colors);

	uint32_t *dev_counts = NULL;
    size_t img_size = dim * dim * sizeof(uint32_t);
    err = hipMalloc(&dev_counts, img_size);
    checkErr(err, "Failed to allocate dev_counts");

	mandel_kernel<<<fraPar.height, fraPar.width>>>(
		
	);

}



